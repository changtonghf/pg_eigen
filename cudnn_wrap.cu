#include <iostream>
#include <array>

#include <Eigen/Core>
#include <unsupported/Eigen/CXX11/Tensor>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CHECK_CUDA(status) \
    do \
    { \
        if (status != hipSuccess) \
        { \
            std::cerr << "CUDA Error: " << hipGetErrorString(status) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUDNN(status) \
    do \
    { \
        if (status != HIPDNN_STATUS_SUCCESS) \
        { \
            std::cerr << "CUDNN Error: " << hipdnnGetErrorString(status) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

template<typename T,hipdnnDataType_t D>
void cudnn_conv2d(T* i1,int* d1,T* k2,int* d2,int* s3,int* p4,T* o5,int* d5)
{
    hipdnnHandle_t handle;
    CHECK_CUDNN(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t xDesc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NHWC, D, d1[0], d1[3], d1[1], d1[2]));

    hipdnnFilterDescriptor_t wDesc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&wDesc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(wDesc, D, HIPDNN_TENSOR_NHWC, d2[3], d2[2], d2[0], d2[1]));

    hipdnnConvolutionDescriptor_t convDesc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc, p4[2], p4[4], s3[1], s3[2], 1, 1, HIPDNN_CROSS_CORRELATION, D));

    hipdnnTensorDescriptor_t yDesc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    int N, C, H, W;
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, xDesc, wDesc, &N, &C, &H, &W));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NHWC, D, N, C, H, W));

    T *dx, *dw, *dy;
    hipMalloc(&dx, d1[0] * d1[1] * d1[2] * d1[3] * sizeof(T));
    hipMalloc(&dw, d2[0] * d2[1] * d2[2] * d2[3] * sizeof(T));
    hipMalloc(&dy, N * C * H * W * sizeof(T));

    Eigen::array<int, 4> n; for (int i=0;i < 4;i++) n[i] = d2[i];
    Eigen::TensorMap<Eigen::Tensor<T, 4, Eigen::RowMajor>> k_(k2, n);
    Eigen::array<ptrdiff_t, 4> f = {3,0,1,2};
    Eigen::Tensor<T, 4, Eigen::RowMajor> k = k_.shuffle(f);

    CHECK_CUDA(hipMemcpy(dx, i1, d1[0] * d1[1] * d1[2] * d1[3] * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dw, k.data(), d2[0] * d2[1] * d2[2] * d2[3] * sizeof(T), hipMemcpyHostToDevice));

    hipdnnConvolutionFwdAlgo_t algo;
    size_t workspaceSize;
    CHECK_CUDNN(hipdnnGetConvolutionForwardAlgorithm(handle, xDesc, wDesc, convDesc, yDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(handle, xDesc, wDesc, convDesc, yDesc, algo, &workspaceSize));

    void *workspace;
    hipMalloc(&workspace, workspaceSize);

    T alpha = 1.0f, beta = 0.0f;

    CHECK_CUDNN(hipdnnConvolutionForward(handle, &alpha, xDesc, dx, wDesc, dw, convDesc, algo, workspace, workspaceSize, &beta, yDesc, dy));
    CHECK_CUDA(hipMemcpy(o5, dy, N * C * H * W * sizeof(T), hipMemcpyDeviceToHost));

    hipFree(dx);
    hipFree(dw);
    hipFree(dy);
    hipFree(workspace);
    hipdnnDestroyTensorDescriptor(xDesc);
    hipdnnDestroyFilterDescriptor(wDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroyTensorDescriptor(yDesc);
    hipdnnDestroy(handle);
}

extern "C" void pg_cudnn_convolve(int oid,void* i1,int n1,int* d1,void* k2,int* d2,int* s3,int* p4,void* o5,int* d5)
{
    if (oid == 700)
    {
        if (n1 == 4)
            cudnn_conv2d<float, HIPDNN_DATA_FLOAT>((float*) i1, d1, (float*) k2, d2, s3, p4, (float*) o5, d5);
    }
    else if (oid == 701)
    {
        if (n1 == 4)
            cudnn_conv2d<double, HIPDNN_DATA_DOUBLE>((double*) i1, d1, (double*) k2, d2, s3, p4, (double*) o5, d5);
    }
}
